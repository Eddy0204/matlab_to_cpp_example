
#include <hip/hip_runtime.h>
#include <iostream>
#include <random>
#include <chrono>
#include <cstdlib>

__global__ void squareCUDA(float* originalMatrix, float* finalMatrix, int total){
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= total){
             return;
    }
    finalMatrix[i] = originalMatrix[i] * originalMatrix[i];
}

int main()
{
    int size = 10000;

    float* originalMatrix = (float*)malloc(size * size * sizeof(float));
    float* finalMatrix = (float*)malloc(size * size * sizeof(float));


    //fill original matrix with random numbers
    for (int i = 0; i < size * size; i++){
        originalMatrix[i] = static_cast<float>(rand()) / RAND_MAX;
    }

    float* originalMatrix_cuda;
    float* finalMatrix_cuda;
    

    int totalElements = size * size;

    hipMalloc(&originalMatrix_cuda, totalElements * sizeof(float));
    hipMalloc(&finalMatrix_cuda, totalElements * sizeof(float));
    hipMemcpy(originalMatrix_cuda, originalMatrix, totalElements * sizeof(float), hipMemcpyHostToDevice);

    //warm up cuda (for accurate timing)
    squareCUDA<<<1, 1>>>(originalMatrix_cuda, finalMatrix_cuda, totalElements);
    hipDeviceSynchronize();

    //Timing start here
    auto start = std::chrono::high_resolution_clock::now();
    
    
    int threadsPerBlock = 256;
    int blocks = (totalElements + threadsPerBlock - 1) / threadsPerBlock;


    squareCUDA<<<blocks, threadsPerBlock>>>(originalMatrix_cuda, finalMatrix_cuda, totalElements);
    hipDeviceSynchronize();

    //end timing
    auto end = std::chrono::high_resolution_clock::now();
    std::chrono::duration<double> elapsed = end - start;

    //copy matrix into cpu
    hipMemcpy(finalMatrix, finalMatrix_cuda, totalElements * sizeof(float), hipMemcpyDeviceToHost);


    //print timing

    //MATLAB time for 10000 x 10000: 0.183926 seconds
    std::cout << "Cuda time for " << size << " x " << size << ": "<< elapsed.count() << " seconds\n";


    hipFree(originalMatrix_cuda);
    hipFree(finalMatrix_cuda);
    free(finalMatrix);
    free(originalMatrix);

    return 0;
}